#include "hip/hip_runtime.h"
#include "vectorMath.hpp"
#include "voxelization.hpp"

#include "IntersectorOctreeGPU.hpp"
#include "pmjSampler.hpp"
#include "renderCommon.hpp"
#include "voxCommon.hpp"
#include "StreamCompaction.hpp"

// method to seperate bits from a given integer 3 positions apart
__device__ inline uint64_t splitBy3( uint32_t a )
{
	uint64_t x = a & 0x1FFFFF;
	x = ( x | x << 32 ) & 0x1f00000000ffff;	 // shift left 32 bits, OR with self, and 00011111000000000000000000000000000000001111111111111111
	x = ( x | x << 16 ) & 0x1f0000ff0000ff;	 // shift left 32 bits, OR with self, and 00011111000000000000000011111111000000000000000011111111
	x = ( x | x << 8 ) & 0x100f00f00f00f00f; // shift left 32 bits, OR with self, and 0001000000001111000000001111000000001111000000001111000000000000
	x = ( x | x << 4 ) & 0x10c30c30c30c30c3; // shift left 32 bits, OR with self, and 0001000011000011000011000011000011000011000011000011000100000000
	x = ( x | x << 2 ) & 0x1249249249249249;
	return x;
}
__device__ inline uint64_t encode2mortonCode_magicbits( uint32_t x, uint32_t y, uint32_t z )
{
	uint64_t answer = 0;
	answer |= splitBy3( x ) | splitBy3( y ) << 1 | splitBy3( z ) << 2;
	return answer;
}

__device__ inline uint32_t getThirdBits( uint64_t m )
{
	const uint64_t masks[6] = { 0x1fffffllu, 0x1f00000000ffffllu, 0x1f0000ff0000ffllu, 0x100f00f00f00f00fllu, 0x10c30c30c30c30c3llu, 0x1249249249249249llu };
	uint64_t x = m & masks[5];
	x = ( x ^ ( x >> 2 ) ) & masks[4];
	x = ( x ^ ( x >> 4 ) ) & masks[3];
	x = ( x ^ ( x >> 8 ) ) & masks[2];
	x = ( x ^ ( x >> 16 ) ) & masks[1];
	x = ( x ^ ( x >> 32 ) ) & masks[0];
	return static_cast<uint32_t>( x );
}
__device__ inline void decodeMortonCode_magicBits( uint64_t morton, uint32_t* x, uint32_t* y, uint32_t* z )
{
	*x = getThirdBits( morton );
	*y = getThirdBits( morton >> 1 );
	*z = getThirdBits( morton >> 2 );
}

template <int NElement, int NThread, class T>
__device__ void clearShared( T* sMem, T value )
{
	for( int i = 0; i < NElement; i += NThread )
	{
		if( i < NElement )
		{
			sMem[i + threadIdx.x] = value;
		}
	}
}

extern "C" __global__ void __launch_bounds__( VOXELIZE_BLOCK_THREADS ) voxCount( const float3* vertices, uint32_t nTriangles, uint32_t* counter, float3 origin, float dps, uint32_t gridRes )
{
	uint32_t iTri = blockIdx.x * blockDim.x + threadIdx.x;

	if( iTri < nTriangles )
	{
		float3 v0 = vertices[iTri * 3];
		float3 v1 = vertices[iTri * 3 + 1];
		float3 v2 = vertices[iTri * 3 + 2];

		bool sixSeparating = true;
		VTContext context( v0, v1, v2, sixSeparating, { origin.x, origin.y, origin.z }, dps, gridRes );
		int2 xrange = context.xRangeInclusive();
		uint32_t nVoxels = 0;
		for( int x = xrange.x; x <= xrange.y; x++ )
		{
			int2 yrange = context.yRangeInclusive( x, dps );
			for( int y = yrange.x; y <= yrange.y; y++ )
			{
				int2 zrange = context.zRangeInclusive( x, y, dps, sixSeparating );
				for( int z = zrange.x; z <= zrange.y; z++ )
				{
					float3 p = context.p( x, y, z, dps );
					if( context.intersect( p ) )
					{
						nVoxels++;
					}
				}
			}
		}
		atomicAdd( counter, nVoxels );
	}
}
extern "C" __global__ void __launch_bounds__( VOXELIZE_BLOCK_THREADS ) voxelize( const float3* vertices, const float3* vcolors, const float3* vemissions, uint32_t nTriangles, uint32_t* counter, float3 origin, float dps, uint32_t gridRes, uint64_t* mortonVoxels, VoxelAttirb* voxelAttribs )
{
	uint32_t iTri = blockIdx.x * blockDim.x + threadIdx.x;

	if( iTri < nTriangles )
	{
		float3 v0 = vertices[iTri * 3];
		float3 v1 = vertices[iTri * 3 + 1];
		float3 v2 = vertices[iTri * 3 + 2];

		float3 c0 = vcolors[iTri * 3];
		float3 c1 = vcolors[iTri * 3 + 1];
		float3 c2 = vcolors[iTri * 3 + 2];

		float3 e0 = vemissions[iTri * 3];
		float3 e1 = vemissions[iTri * 3 + 1];
		float3 e2 = vemissions[iTri * 3 + 2];

		bool sixSeparating = true;
		VTContext context( v0, v1, v2, sixSeparating, { origin.x, origin.y, origin.z }, dps, gridRes );
		int2 xrange = context.xRangeInclusive();
		uint32_t nVoxels = 0;
		for( int x = xrange.x; x <= xrange.y; x++ )
		{
			int2 yrange = context.yRangeInclusive( x, dps );
			for( int y = yrange.x; y <= yrange.y; y++ )
			{
				int2 zrange = context.zRangeInclusive( x, y, dps, sixSeparating );
				for( int z = zrange.x; z <= zrange.y; z++ )
				{
					float3 p = context.p( x, y, z, dps );
					if( context.intersect( p ) )
					{
						nVoxels++;
					}
				}
			}
		}

		uint32_t dstLocation = atomicAdd( counter, nVoxels );
		nVoxels = 0;

		for( int x = xrange.x; x <= xrange.y; x++ )
		{
			int2 yrange = context.yRangeInclusive( x, dps );
			for( int y = yrange.x; y <= yrange.y; y++ )
			{
				int2 zrange = context.zRangeInclusive( x, y, dps, sixSeparating );
				for( int z = zrange.x; z <= zrange.y; z++ )
				{
					float3 p = context.p( x, y, z, dps );
					if( context.intersect( p ) )
					{
						int3 c = context.i( x, y, z );
						mortonVoxels[dstLocation + nVoxels] = encode2mortonCode_magicbits( c.x, c.y, c.z );

						float3 bc = closestBarycentricCoordinateOnTriangle( v0, v1, v2, p );
						float3 bColor = bc.x * c1 + bc.y * c2 + bc.z * c0;
						float3 bEmission = bc.x * e1 + bc.y * e2 + bc.z * e0;

						voxelAttribs[dstLocation + nVoxels].color = {
							(uint8_t)( bColor.x * 255.0f + 0.5f ),
							(uint8_t)( bColor.y * 255.0f + 0.5f ),
							(uint8_t)( bColor.z * 255.0f + 0.5f ), 255 };
						voxelAttribs[dstLocation + nVoxels].emission = {
							(uint8_t)( bEmission.x * 255.0f + 0.5f ),
							(uint8_t)( bEmission.y * 255.0f + 0.5f ),
							(uint8_t)( bEmission.z * 255.0f + 0.5f ), 255 };

						nVoxels++;
					}
				}
			}
		}
	}
}

extern "C" __global__ void unique( const uint64_t* inputMortonVoxels, uint64_t* outputMortonVoxels, const VoxelAttirb* inputVoxelAttribs, VoxelAttirb* outputVoxelAttribs, uint32_t totalDumpedVoxels, StreamCompaction streamCompaction )
{
	streamCompaction.filter<UNIQUE_BLOCK_SIZE /*ITEMS_PER_BLOCK*/, UNIQUE_BLOCK_THREADS /*BLOCK_DIM*/>(
		[&]( int srcIndex )
		{
			if( srcIndex < totalDumpedVoxels )
			{
				return srcIndex == 0 || inputMortonVoxels[srcIndex - 1] != inputMortonVoxels[srcIndex];
			}
			return false;
		},
		[&]( int srcIndex, int dstIndex )
		{
			uint64_t morton = inputMortonVoxels[srcIndex];
			outputMortonVoxels[dstIndex] = morton;

			int R = 0;
			int G = 0;
			int B = 0;
			int Re = 0;
			int Ge = 0;
			int Be = 0;
			int n = 0;
			for( int j = srcIndex; j < totalDumpedVoxels && inputMortonVoxels[j] == morton; j++ )
			{
				R += inputVoxelAttribs[j].color.x;
				G += inputVoxelAttribs[j].color.y;
				B += inputVoxelAttribs[j].color.z;
				Re += inputVoxelAttribs[j].emission.x;
				Ge += inputVoxelAttribs[j].emission.y;
				Be += inputVoxelAttribs[j].emission.z;
				n++;
			}
			uchar4 meanColor = {
				(uint8_t)( R / n ),
				(uint8_t)( G / n ),
				(uint8_t)( B / n ),
				255 };
			uchar4 meanEmission = {
				(uint8_t)( Re / n ),
				(uint8_t)( Ge / n ),
				(uint8_t)( Be / n ),
				255 };
			outputVoxelAttribs[dstIndex].color = meanColor;
			outputVoxelAttribs[dstIndex].emission = meanEmission;
		} 
	);
}

extern "C" __global__ void octreeTaskInit( const uint64_t* inputMortonVoxels, uint32_t numberOfVoxels, OctreeTask* outputOctreeTasks, uint32_t* taskCounters, uint32_t gridRes )
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if( i < numberOfVoxels )
	{
		uint64_t mortonL = inputMortonVoxels[max( i - 1, 0 )];
		uint64_t mortonR = inputMortonVoxels[i];

		outputOctreeTasks[i].morton = mortonR;
		outputOctreeTasks[i].child = 0xFFFFFFFF;
		outputOctreeTasks[i].numberOfVoxels = 1;

		int iteration = 0;
		while( 1 < ( gridRes >> iteration ) )
		{
			if( i == 0 || mortonL >> ( 3 * ( iteration + 1 ) ) != mortonR >> ( 3 * ( iteration + 1 ) ) )
			{
				atomicInc( &taskCounters[iteration], 0xFFFFFFFF );
			}
			iteration++;
		}
	}
}

extern "C" __global__ void bottomUpOctreeBuild(
	int iteration,
	const OctreeTask* inputOctreeTasks, uint32_t nInput,
	OctreeTask* outputOctreeTasks,
	OctreeNode* outputOctreeNodes, uint32_t* nOutputNodes,
	uint32_t* lpBuffer, uint32_t lpSize,
	StreamCompaction streamCompaction )
{
	streamCompaction.filter<BOTTOM_UP_BLOCK_SIZE /*ITEMS_PER_BLOCK*/, BOTTOM_UP_BLOCK_THREADS /*BLOCK_DIM*/>(
		[&]( int srcIndex )
		{
			if( srcIndex < nInput )
			{
				return srcIndex == 0 || inputOctreeTasks[srcIndex - 1].getMortonParent() != inputOctreeTasks[srcIndex].getMortonParent();
			}
			return false;
		},
		[&]( int srcIndex, int dstIndex )
		{
			uint8_t mask = 0;
			
			uint32_t children[8];
			uint32_t nVoxelsPSum[8];
			for( int j = 0; j < 8; j++ )
			{
				children[j] = 0xFFFFFFFF;
				nVoxelsPSum[j] = 0;
			}

			// set child
			uint64_t mortonParent = inputOctreeTasks[srcIndex].getMortonParent();
			for( int j = srcIndex; j < nInput && inputOctreeTasks[j].getMortonParent() == mortonParent; j++ )
			{
				uint32_t space = inputOctreeTasks[j].morton & 0x7;
				mask |= ( 1 << space ) & 0xFF;
				children[space] = inputOctreeTasks[j].child;
				nVoxelsPSum[space] = inputOctreeTasks[j].numberOfVoxels;
			}

			// prefix scan exclusive
			uint32_t numberOfVoxels = 0;
			for( int j = 0; j < 8; j++ )
			{
				uint32_t c = nVoxelsPSum[j];
				nVoxelsPSum[j] = numberOfVoxels;
				numberOfVoxels += c;
			}

#if !defined( ENABLE_GPU_DAG )
			// Non DAG
			uint32_t nodeIndex = atomicInc( nOutputNodes, 0xFFFFFFFF );
			outputOctreeNodes[nodeIndex].mask = mask;
			for( int j = 0; j < 8; j++ )
			{
				outputOctreeNodes[nodeIndex].children[j] = children[j];
				outputOctreeNodes[nodeIndex].nVoxelsPSum[j] = nVoxelsPSum[j];
			}

			outputOctreeTasks[dstIndex].morton = mortonParent;
			outputOctreeTasks[dstIndex].child = nodeIndex;
			outputOctreeTasks[dstIndex].numberOfVoxels = numberOfVoxels;
#else
			// DAG
			uint32_t nodeIndex = 0xFFFFFFFF;

			MurmurHash32 h( 0 );
			h.combine( mask );
			for( int i = 0; i < 8; i++ )
				h.combine( children[i] );
			uint32_t home = h.getHash() % lpSize;

			bool done = false;
#if defined( ITS )
			uint32_t active = __activemask();
			for( int i = 0; __all_sync( active, done ) == false; i++, __syncwarp( active ) )
#else
			for( int i = 0; __all( done ) == false; i++ )
#endif
			{
				if( done )
				{
					continue;
				}

				int location = ( home + i ) % lpSize;
				uint32_t v = atomicCAS( &lpBuffer[location], 0, LP_LOCK );

				__threadfence();

				if( v == 0 ) // succeeded to lock
				{
					nodeIndex = atomicInc( nOutputNodes, 0xFFFFFFFF );
					outputOctreeNodes[nodeIndex].mask = mask;
					for( int j = 0; j < 8; j++ )
					{
						outputOctreeNodes[nodeIndex].children[j] = children[j];
						outputOctreeNodes[nodeIndex].nVoxelsPSum[j] = nVoxelsPSum[j];
					}

					__threadfence();

					atomicExch( &lpBuffer[location], nodeIndex | LP_OCCUPIED_BIT );

					done = true;
				}
				else if( v == LP_LOCK ) // someone is locking it
				{
					i--; // try again
				}
				else // existing item
				{
					uint32_t otherNodeIndex = v & LP_VALUE_BIT;
					bool isEqual = outputOctreeNodes[otherNodeIndex].mask == mask;
					if( isEqual )
					{
						for( int j = 0; j < 8; j++ )
						{
							if( outputOctreeNodes[otherNodeIndex].children[j] != children[j] )
							{
								isEqual = false;
								break;
							}
						}
					}
					if( isEqual )
					{
						nodeIndex = otherNodeIndex;

						done = true;
					}
				}
			}

			outputOctreeTasks[dstIndex].morton = mortonParent;
			outputOctreeTasks[dstIndex].child = nodeIndex;
			outputOctreeTasks[dstIndex].numberOfVoxels = numberOfVoxels;
#endif
		} 
	);
}

extern "C" __global__ void embedMasks( OctreeNode *nodes, uint32_t numberOfNodes )
{
	uint32_t nodeIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if( numberOfNodes <= nodeIndex )
	{
		return;
	}
	
	embedMask( nodes, nodeIndex );
}

extern "C" __global__ void render(
	uchar4* frameBuffer, int2 resolution,
	uint32_t* taskCounter, StackElement* stackBuffer,
	CameraPinhole pinhole,
	IntersectorOctreeGPU intersector,
	int showVertexColor )
{
	__shared__ uint32_t taskIdx;

	StackElement* stack = stackBuffer + blockIdx.x * 32 * blockDim.x + threadIdx.x * 32;

	for( ;; )
	{
		if( threadIdx.x == 0 )
		{
			taskIdx = atomicInc( taskCounter, 0xFFFFFFFF );
		}
		__syncthreads();

		uint32_t pixelIdx = taskIdx * blockDim.x + threadIdx.x;
		if( resolution.x * resolution.y <= pixelIdx )
		{
			break;
		}

		uint32_t x = pixelIdx % resolution.x;
		uint32_t y = pixelIdx / resolution.x;

		float3 ro, rd;
		pinhole.shoot( &ro, &rd, x, y, 0.5f, 0.5f, resolution.x, resolution.y );

		float t = MAX_FLOAT;
		int nMajor;
		uint32_t vIndex = 0;
		intersector.intersect( stack, ro, rd, &t, &nMajor, &vIndex, false /* isShadowRay */ );
		uchar4 colorOut = { 0, 0, 0, 255 };
		if( t != MAX_FLOAT )
		{
			if( showVertexColor )
			{
				colorOut = intersector.getVoxelColor( vIndex );
			}
			else
			{
				float3 hitN = getHitN( nMajor, rd );
				float3 color = ( hitN + float3{ 1.0f, 1.0f, 1.0f } ) * 0.5f;
				colorOut = { 
					(uint8_t)( 255 * color.x + 0.5f ), 
					(uint8_t)( 255 * color.y + 0.5f ), 
					(uint8_t)( 255 * color.z + 0.5f ), 
					255 };
			}
		}
		frameBuffer[y * resolution.x + x] = colorOut;
	}
}

extern "C" __global__ void HDRIstoreImportance( const float4* pixels, int2 resolution, double* sat, int cosWeighted, float3 axis )
{
	uint32_t pixelX = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t pixelY = blockIdx.y * blockDim.y + threadIdx.y;
	if( resolution.x <= pixelX || resolution.y <= pixelY )
	{
		return;
	}

	uint32_t pixelIdx = pixelY * resolution.x + pixelX;
	float dTheta = PI / (float)resolution.y;
	float dPhi = 2.0f * PI / (float)resolution.x;
	float theta = pixelY * dTheta;

	// dH = cos( theta ) - cos( theta + dTheta )
	//    = 2 sin( dTheta / 2 ) sin( dTheta / 2 + theta )
	float dH = 2.0f * INTRIN_SIN( dTheta * 0.5f ) * INTRIN_SIN( dTheta * 0.5f + theta );
	float dW = dPhi;
	float sr = dH * dW;
	float4 color = pixels[pixelIdx];

	float w = 1.0f;
	if( cosWeighted )
	{
		float sY = mix( INTRIN_COS( theta ), INTRIN_COS( theta + dTheta ), 0.5f );
		float phi = dPhi * ( (float)pixelX + 0.5f ) + PI;
		float sX = INTRIN_COS( phi );
		float sZ = INTRIN_SIN( phi );

		float sinTheta = INTRIN_SQRT( ss_max( 1.0f - sY * sY, 0.0f ) );
		float3 dirCenter = {
			sX * sinTheta,
			sY,
			sZ * sinTheta,
		};
		w = ss_max( dot( axis, dirCenter ), 0.0f );
	}

	sat[pixelIdx] = luminance( color ) * sr * w;
}

template <class T, int NThreads>
__device__ inline T prefixSumInclusive( T prefix, T* sMemIO )
{
	for( uint32_t offset = 1; offset < NThreads; offset <<= 1 )
	{
		T x = sMemIO[threadIdx.x];

		if( offset <= threadIdx.x )
		{
			x += sMemIO[threadIdx.x - offset];
		}

		__syncthreads();

		sMemIO[threadIdx.x] = x;

		__syncthreads();
	}
	T sum = sMemIO[NThreads - 1];

	__syncthreads();

	sMemIO[threadIdx.x] += prefix;

	__syncthreads();

	return sum;
}

#define SAT_BLOCK_SIZE 512

extern "C" __global__ void buildSATh( int2 resolution, double* sat )
{
	__shared__ double s_mem[SAT_BLOCK_SIZE];
	int Y = blockIdx.x;

	double prefix = 0.0;
	for( int i = 0; i < resolution.x; i += SAT_BLOCK_SIZE )
	{
		int X = i + threadIdx.x;
		s_mem[threadIdx.x] = X < resolution.x ? sat[Y * resolution.x + X] : 0.0;

		__syncthreads();

		prefix += prefixSumInclusive<double, SAT_BLOCK_SIZE>( prefix, s_mem );

		if( X < resolution.x )
		{
			sat[Y * resolution.x + X] = s_mem[threadIdx.x];
		}
	}
}
extern "C" __global__ void buildSATv( int2 resolution, double* sat )
{
	__shared__ double s_mem[SAT_BLOCK_SIZE];
	int X = blockIdx.x;

	double prefix = 0.0;
	for( int i = 0; i < resolution.y; i += SAT_BLOCK_SIZE )
	{
		int Y = i + threadIdx.x;
		s_mem[threadIdx.x] = Y < resolution.y ? sat[Y * resolution.x + X] : 0.0;

		__syncthreads();

		prefix += prefixSumInclusive<double, SAT_BLOCK_SIZE>( prefix, s_mem );

		if( Y < resolution.y )
		{
			sat[Y * resolution.x + X] = s_mem[threadIdx.x];
		}
	}
}

extern "C" __global__ void buildSAT2u32( uint32_t* satU32, double* satF64, int n )
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	double sum = satF64[n - 1];
	if( i < n )
	{
		satU32[i] = (uint32_t)( satF64[i] / ( sum ) * (double)0xFFFFFFFFu );
	}
}

extern "C" __global__ void __launch_bounds__( RENDER_NUMBER_OF_THREAD ) renderPT(
	int iteration,
	float4* frameBuffer, int2 resolution,
	CameraPinhole pinhole,
	IntersectorOctreeGPU intersector,
	DynamicAllocatorGPU<StackElement> stackAllocator,
	HDRI hdri,
	PMJSampler pmj )
{
	uint32_t stackHandle;
	StackElement* stack = stackAllocator.acquire( &stackHandle );

	__shared__ float localPixelValueXs[RENDER_NUMBER_OF_THREAD];
	__shared__ float localPixelValueYs[RENDER_NUMBER_OF_THREAD];
	__shared__ float localPixelValueZs[RENDER_NUMBER_OF_THREAD];
	localPixelValueXs[threadIdx.x] = 0.0f;
	localPixelValueYs[threadIdx.x] = 0.0f;
	localPixelValueZs[threadIdx.x] = 0.0f;

	__syncthreads();

	const int nBatchSpp = 16;

	for( int i = 0; i < nBatchSpp * RENDER_NUMBER_OF_THREAD; i += RENDER_NUMBER_OF_THREAD )
	{
		uint32_t taskIdx = i + threadIdx.x;
		uint32_t localPixel = taskIdx / nBatchSpp;
		uint32_t localSpp = taskIdx % nBatchSpp;

		uint32_t pixelIdx = blockIdx.x * blockDim.x + localPixel;
		uint32_t x = pixelIdx % resolution.x;
		uint32_t y = pixelIdx / resolution.x;
		uint32_t spp = iteration * nBatchSpp + localSpp;
		if( blockDim.x <= localPixel || resolution.x <= x || resolution.y <= y )
		{
			break;
		}

		MurmurHash32 hash( 0 );
		hash.combine( pixelIdx );

#if defined( USE_PMJ )
		int dim = 0;
		uint32_t stream = hash.getHash();
#define SAMPLE_2D() pmj.sample2d( spp, dim++, stream )
#else
		hash.combine( spp );
		PCG32 rng;
		rng.setup( 0, hash.getHash() );
#define SAMPLE_2D() float2{ uniformf( rng.nextU32() ), uniformf( rng.nextU32() ) }
#endif

		float2 cam_u01 = SAMPLE_2D();
		float3 ro, rd;
		// pinhole.shoot( &ro, &rd, x, y, cam_u01.x, cam_u01.y, resolution.x, resolution.y );

		float2 lens_u01 = SAMPLE_2D();
		pinhole.shootThinLens( &ro, &rd, x, y, cam_u01.x, cam_u01.y, resolution.x, resolution.y, lens_u01.x, lens_u01.y );

		float3 T = { 1.0f, 1.0f, 1.0f };
		float3 L = {};

		float t = MAX_FLOAT;
		int nMajor;
		uint32_t vIndex = 0;
		intersector.intersect( stack, ro, rd, &t, &nMajor, &vIndex, false /* isShadowRay */ );

		// Primary emissions:
		if( t == MAX_FLOAT )
		{
			// float I = ss_max( normalize( rd ).y, 0.0f ) * 3.0f;
			// float3 env = { I, I, I };
			float3 env = hdri.sampleNearest( rd, true );
			L += T * env;
		}
		else
		{
			float3 Le = intersector.getVoxelEmission( vIndex, false );
			L += T * Le;
		}

		for( int depth = 0; depth < 8 && t != MAX_FLOAT; depth++ )
		{
			// float3 R = linearReflectance( intersector.getVoxelColor( vIndex ) );
			float3 R = rawReflectance( intersector.getVoxelColor( vIndex ) );
			float3 hitN = getHitN( nMajor, rd );
			float3 hitP = ro + rd * t;

			if( hdri.isEnabled() )
			{ // Explicit
				float2 u01 = SAMPLE_2D();
				float2 u23 = SAMPLE_2D();

				float3 dir;
				float3 emissive;
				float p;
				hdri.importanceSample( &dir, &emissive, &p, hitN, true, u01.x, u01.y, u23.x, u23.y );

				// no self intersection
				float t = MAX_FLOAT;
				int nMajor;
				uint32_t vIndex = 0;
				intersector.intersect( stack, hitP, dir, &t, &nMajor, &vIndex, true /* isShadowRay */ );
				if( t == MAX_FLOAT )
				{
					L += T * ( R / PI ) * ss_max( dot( hitN, dir ), 0.0f ) * emissive / p;
				}
			}

			float2 u01 = SAMPLE_2D();
			float2 u23 = SAMPLE_2D();

			T *= R;
			float3 dir = sampleLambertian( u01.x, u01.y, hitN );

			ro = hitP; // no self intersection
			rd = dir;

			t = MAX_FLOAT;
			intersector.intersect( stack, ro, rd, &t, &nMajor, &vIndex, false /* isShadowRay */ );

			if( t != MAX_FLOAT )
			{
				float3 Le = intersector.getVoxelEmission( vIndex, true );
				L += T * Le;
			}
		}

#undef SAMPLE_2D
		atomicAdd( &localPixelValueXs[localPixel], L.x );
		atomicAdd( &localPixelValueYs[localPixel], L.y );
		atomicAdd( &localPixelValueZs[localPixel], L.z );
	}

	__syncthreads();

	uint32_t pixelIdx = blockIdx.x * blockDim.x + threadIdx.x;
	frameBuffer[pixelIdx].x += localPixelValueXs[threadIdx.x];
	frameBuffer[pixelIdx].y += localPixelValueYs[threadIdx.x];
	frameBuffer[pixelIdx].z += localPixelValueZs[threadIdx.x];
	frameBuffer[pixelIdx].w += (float)nBatchSpp;

	stackAllocator.release( stackHandle );
}

extern "C" __global__ void renderResolve( uchar4* frameBufferU8, const float4* frameBufferF32, int n )
{
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if( i < n )
	{
		float4 value = frameBufferF32[i];
		int r = (int)( 255 * INTRIN_POW( value.x / value.w, 1.0f / 2.2f ) + 0.5f );
		int g = (int)( 255 * INTRIN_POW( value.y / value.w, 1.0f / 2.2f ) + 0.5f );
		int b = (int)( 255 * INTRIN_POW( value.z / value.w, 1.0f / 2.2f ) + 0.5f );
		uchar4 colorOut = {
			(uint8_t)min( r, 255 ),
			(uint8_t)min( g, 255 ),
			(uint8_t)min( b, 255 ),
			255 };
		frameBufferU8[i] = colorOut;
	}
}
