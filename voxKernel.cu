#include "hip/hip_runtime.h"
#include "vectorMath.hpp"
#include "voxelization.hpp"

#include "voxCommon.hpp"
#include "IntersectorOctreeGPU.hpp"
#include "renderCommon.hpp"
#include "pmjSampler.hpp"

// method to seperate bits from a given integer 3 positions apart
__device__ inline uint64_t splitBy3( uint32_t a )
{
	uint64_t x = a & 0x1FFFFF;
	x = ( x | x << 32 ) & 0x1f00000000ffff;	 // shift left 32 bits, OR with self, and 00011111000000000000000000000000000000001111111111111111
	x = ( x | x << 16 ) & 0x1f0000ff0000ff;	 // shift left 32 bits, OR with self, and 00011111000000000000000011111111000000000000000011111111
	x = ( x | x << 8 ) & 0x100f00f00f00f00f; // shift left 32 bits, OR with self, and 0001000000001111000000001111000000001111000000001111000000000000
	x = ( x | x << 4 ) & 0x10c30c30c30c30c3; // shift left 32 bits, OR with self, and 0001000011000011000011000011000011000011000011000011000100000000
	x = ( x | x << 2 ) & 0x1249249249249249;
	return x;
}
__device__ inline uint64_t encode2mortonCode_magicbits( uint32_t x, uint32_t y, uint32_t z )
{
	uint64_t answer = 0;
	answer |= splitBy3( x ) | splitBy3( y ) << 1 | splitBy3( z ) << 2;
	return answer;
}

template <int NElement, int NThread, class T>
__device__ void clearShared( T* sMem, T value )
{
	for( int i = 0; i < NElement; i += NThread )
	{
		if( i < NElement )
		{
			sMem[i + threadIdx.x] = value;
		}
	}
}

extern "C" __global__ void voxCount( const float3 *vertices, const float3 *vcolors, uint32_t nTriangles, uint32_t* counter, float3 origin, float dps, uint32_t gridRes )
{
    uint32_t iTri = blockIdx.x * blockDim.x + threadIdx.x;

    if( iTri < nTriangles )
    {
        float3 v0 = vertices[iTri * 3];
        float3 v1 = vertices[iTri * 3 + 1];
        float3 v2 = vertices[iTri * 3 + 2];

        bool sixSeparating = true;
        VTContext context( v0, v1, v2, sixSeparating, { origin.x, origin.y, origin.z }, dps, gridRes );
        int2 xrange = context.xRangeInclusive();
        uint32_t nVoxels = 0;
        for( int x = xrange.x; x <= xrange.y; x++ )
        {
            int2 yrange = context.yRangeInclusive( x, dps );
            for( int y = yrange.x; y <= yrange.y; y++ )
            {
                int2 zrange = context.zRangeInclusive( x, y, dps, sixSeparating );
                for( int z = zrange.x; z <= zrange.y; z++ )
                {
                    float3 p = context.p( x, y, z, dps );
                    if( context.intersect( p ) )
                    {
                        nVoxels++;
                    }
                }
            }
        }
        atomicAdd( counter, nVoxels );
    }
}
extern "C" __global__ void voxelize( const float3 *vertices, const float3 *vcolors, uint32_t nTriangles, uint32_t* counter, float3 origin, float dps, uint32_t gridRes, uint64_t* mortonVoxels, uchar4* voxelColors )
{
    uint32_t iTri = blockIdx.x * blockDim.x + threadIdx.x;

    if( iTri < nTriangles )
    {
        float3 v0 = vertices[iTri * 3];
        float3 v1 = vertices[iTri * 3 + 1];
        float3 v2 = vertices[iTri * 3 + 2];

        float3 c0 = vcolors[iTri * 3];
        float3 c1 = vcolors[iTri * 3 + 1];
        float3 c2 = vcolors[iTri * 3 + 2];

        bool sixSeparating = true;
        VTContext context( v0, v1, v2, sixSeparating, { origin.x, origin.y, origin.z }, dps, gridRes );
        int2 xrange = context.xRangeInclusive();
        uint32_t nVoxels = 0;
        for( int x = xrange.x; x <= xrange.y; x++ )
        {
            int2 yrange = context.yRangeInclusive( x, dps );
            for( int y = yrange.x; y <= yrange.y; y++ )
            {
                int2 zrange = context.zRangeInclusive( x, y, dps, sixSeparating );
                for( int z = zrange.x; z <= zrange.y; z++ )
                {
                    float3 p = context.p( x, y, z, dps );
                    if( context.intersect( p ) )
                    {
                        nVoxels++;
                    }
                }
            }
        }
        uint32_t dstLocation = atomicAdd( counter, nVoxels );
        nVoxels = 0;

        for( int x = xrange.x; x <= xrange.y; x++ )
        {
            int2 yrange = context.yRangeInclusive( x, dps );
            for( int y = yrange.x; y <= yrange.y; y++ )
            {
                int2 zrange = context.zRangeInclusive( x, y, dps, sixSeparating );
                for( int z = zrange.x; z <= zrange.y; z++ )
                {
                    float3 p = context.p( x, y, z, dps );
                    if( context.intersect( p ) )
                    {
                        int3 c = context.i( x, y, z );
                        mortonVoxels[dstLocation + nVoxels] = encode2mortonCode_magicbits( c.x, c.y, c.z );

                        float3 bc = closestBarycentricCoordinateOnTriangle( v0, v1, v2, p );
                        float3 bColor = bc.x * c1 + bc.y * c2 + bc.z * c0;

						voxelColors[dstLocation + nVoxels] = { 
                            (uint8_t)( bColor.x * 255.0f + 0.5f ), 
                            (uint8_t)( bColor.y * 255.0f + 0.5f ), 
                            (uint8_t)( bColor.z * 255.0f + 0.5f ), 255 };
                        
                        nVoxels++;
                    }
                }
            }
        }
    }
}

#define UNIQUE_BLOCK_SIZE 2048
#define UNIQUE_BLOCK_THREADS 64
#define UNIQUE_NUMBER_OF_ITERATION ( UNIQUE_BLOCK_SIZE / UNIQUE_BLOCK_THREADS )

template <int BLOCK_SIZE>
struct StreamCompaction64
{
    enum { 
        THREADS = 64,
        NUMBER_OF_STEPS = BLOCK_SIZE / THREADS
    };
	uint32_t gp;
	uint64_t leaderMasks[NUMBER_OF_STEPS];

    __device__ void init()
    {
		clearShared<NUMBER_OF_STEPS, THREADS, uint64_t>( leaderMasks, 0 );
		__syncthreads();
    }
	__device__ int steps() const { return NUMBER_OF_STEPS; }
	__device__ uint32_t itemIndex( int step ) const { return blockIdx.x * BLOCK_SIZE + step * THREADS + threadIdx.x; }
	__device__ void vote( int step )
    {
		atomicOr( &leaderMasks[step], 1llu << threadIdx.x );
    }

    // return global prefix
    __device__ uint32_t synchronize( uint64_t* iterator )
    {
		__syncthreads();

		if( threadIdx.x == 0 )
		{
			uint32_t prefix = 0;
			for( int i = 0; i < NUMBER_OF_STEPS; i++ )
			{
				prefix += __popcll( leaderMasks[i] );
			}

			uint64_t expected;
			uint64_t cur = *iterator;
			uint32_t globalPrefix = cur & 0xFFFFFFFF;
			do
			{
				expected = (uint64_t)globalPrefix + ( (uint64_t)( blockIdx.x ) << 32 );
				uint64_t newValue = (uint64_t)globalPrefix + prefix | ( (uint64_t)( blockIdx.x + 1 ) << 32 );
				cur = atomicCAS( iterator, expected, newValue );
				globalPrefix = cur & 0xFFFFFFFF;

			} while( cur != expected );

			gp = globalPrefix;
		}

		__syncthreads();

		return gp;
    }

	// return destination. If it is not voted, return -1
	__device__ uint32_t destination( int step, uint32_t* globalPrefix ) const
    {
		uint64_t mask = leaderMasks[step];
		bool voted = ( mask & ( 1llu << threadIdx.x ) ) != 0;
		uint64_t lowerMask = ( 1llu << threadIdx.x ) - 1;
		uint32_t offset = __popcll( mask & lowerMask );
        uint32_t d = *globalPrefix + offset;
		*globalPrefix += __popcll( mask );
		return voted ? d : -1;
    }
};

extern "C" __global__ void unique( const uint64_t* inputMortonVoxels, uint64_t* outputMortonVoxels, const uchar4* inputVoxelColors, uchar4* outputVoxelColors, uint32_t totalDumpedVoxels, uint64_t *iterator )
{
	__shared__ StreamCompaction64<UNIQUE_BLOCK_SIZE> streamCompaction;
	streamCompaction.init();

	for (int i = 0; i < streamCompaction.steps(); i++)
	{
		uint32_t itemIndex = streamCompaction.itemIndex( i );
		if( itemIndex < totalDumpedVoxels )
		{
			bool leader = itemIndex == 0 || inputMortonVoxels[itemIndex - 1] != inputMortonVoxels[itemIndex];
			if( leader )
			{
				streamCompaction.vote( i );
			}
		}
	}

	uint32_t globalPrefix = streamCompaction.synchronize( iterator );

	for( int i = 0; i < streamCompaction.steps(); i++ )
	{
		uint32_t itemIndex = streamCompaction.itemIndex( i );
		uint32_t d = streamCompaction.destination( i, &globalPrefix );
		if( d != -1 ) // voted
		{
			uint64_t morton = inputMortonVoxels[itemIndex];
			outputMortonVoxels[d] = morton;

			int R = 0;
			int G = 0;
			int B = 0;
			int n = 0;
			for( int j = itemIndex; j < totalDumpedVoxels && inputMortonVoxels[j] == morton; j++ )
			{
				R += inputVoxelColors[j].x;
				G += inputVoxelColors[j].y;
				B += inputVoxelColors[j].z;
				n++;
			}
			uchar4 meanColor = {
				(uint8_t)( R / n ),
				(uint8_t)( G / n ),
				(uint8_t)( B / n ),
				255 };

			outputVoxelColors[d] = meanColor;
		}
	}
}


__device__ uint64_t g_octreeIterator0;
__device__ uint64_t g_octreeIterator1;

extern "C" __global__ void octreeTaskInit( const uint64_t* inputMortonVoxels, uint32_t numberOfVoxels, OctreeTask* outputOctreeTasks, uint32_t* taskCounters, uint32_t gridRes )
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i < numberOfVoxels )
    {
		uint64_t mortonL = inputMortonVoxels[max( i - 1, 0)];
		uint64_t mortonR = inputMortonVoxels[i];

		outputOctreeTasks[i].morton = mortonR;
		outputOctreeTasks[i].child = -1;
		outputOctreeTasks[i].numberOfVoxels = 1;

		int iteration = 0;
		while( 1 < ( gridRes >> iteration ) )
		{
			if( i == 0 || mortonL >> ( 3 * ( iteration + 1 ) ) != mortonR >> ( 3 * ( iteration + 1 ) ) )
			{
				atomicInc( &taskCounters[iteration], 0xFFFFFFFF );
			}
			iteration++;
		}
    }

    if( i == 0 )
    {
		g_octreeIterator0 = 0;
    }
}

#define BOTTOM_UP_BLOCK_SIZE 4096
extern "C" __global__ void bottomUpOctreeBuild( 
	int iteration,
	const OctreeTask* inputOctreeTasks, uint32_t nInput, 
	OctreeTask* outputOctreeTasks, 
	OctreeNode* outputOctreeNodes, uint32_t* nOutputNodes,
	uint32_t* lpBuffer, uint32_t lpSize )
{
	__shared__ StreamCompaction64<BOTTOM_UP_BLOCK_SIZE> streamCompaction;
	streamCompaction.init();

	for( int i = 0; i < streamCompaction.steps(); i++ )
	{
		uint32_t itemIndex = streamCompaction.itemIndex( i );
		if( itemIndex < nInput )
		{
			bool leader = itemIndex == 0 || inputOctreeTasks[itemIndex - 1].getMortonParent() != inputOctreeTasks[itemIndex].getMortonParent();
			if( leader )
			{
				streamCompaction.vote( i );
			}
		}
	}

	uint32_t globalPrefix = streamCompaction.synchronize( iteration % 2 == 0 ? &g_octreeIterator0 : &g_octreeIterator1 );

	for( int i = 0; i < streamCompaction.steps(); i++ )
	{
		uint32_t itemIndex = streamCompaction.itemIndex( i );
		uint32_t d = streamCompaction.destination( i, &globalPrefix );

		uint8_t mask = 0;
		uint32_t numberOfVoxels = 0;
		uint32_t children[8];
		uint32_t nVoxelsPSum[8];
		for( int j = 0; j < 8; j++ )
		{
			children[j] = -1;
			nVoxelsPSum[j] = 0;
		}

		if( d != -1 ) // voted
		{
			// set child
			uint64_t mortonParent = inputOctreeTasks[itemIndex].getMortonParent();
			for( int j = itemIndex; j < nInput && inputOctreeTasks[j].getMortonParent() == mortonParent; j++ )
			{
				uint32_t space = inputOctreeTasks[j].morton & 0x7;
				mask |= ( 1 << space ) & 0xFF;
				children[space] = inputOctreeTasks[j].child;
				nVoxelsPSum[space] = inputOctreeTasks[j].numberOfVoxels;
			}

			// prefix scan exclusive
			for( int j = 0; j < 8; j++ )
			{
				uint32_t c = nVoxelsPSum[j];
				nVoxelsPSum[j] = numberOfVoxels;
				numberOfVoxels += c;
			}
			
			// Non DAG
#if !defined( ENABLE_GPU_DAG )
			uint32_t nodeIndex = atomicInc( nOutputNodes, 0xFFFFFFFF );
			outputOctreeNodes[nodeIndex].mask = mask;
			for( int j = 0; j < 8; j++ )
			{
				outputOctreeNodes[nodeIndex].children[j] = children[j];
				outputOctreeNodes[nodeIndex].nVoxelsPSum[j] = nVoxelsPSum[j];
			}

			outputOctreeTasks[d].morton = mortonParent;
			outputOctreeTasks[d].child = nodeIndex;
			outputOctreeTasks[d].numberOfVoxels = numberOfVoxels;
#endif
		}

#if defined( ENABLE_GPU_DAG )
		uint32_t nodeIndex = -1;

		MurmurHash32 h( 0 );
		h.combine( mask );
		for( int i = 0; i < 8; i++ )
			h.combine( children[i] );
		uint32_t home = h.getHash() % lpSize;

		bool done = d == -1;
#if defined( ITS )
		__syncwarp();
		for( int i = 0; __all_sync( 0xFFFFFFFF, done ) == false; i++ )
#else
		for( int i = 0; __all( done ) == false; i++ )
#endif
		{
			if( done )
			{
				continue;
			}

			int location = ( home + i ) % lpSize;
			uint32_t v = atomicCAS( &lpBuffer[location], 0, LP_LOCK );

			__threadfence();

			if( v == 0 ) // succeeded to lock
			{
				nodeIndex = atomicInc( nOutputNodes, 0xFFFFFFFF );
				outputOctreeNodes[nodeIndex].mask = mask;
				for( int j = 0; j < 8; j++ )
				{
					outputOctreeNodes[nodeIndex].children[j] = children[j];
					outputOctreeNodes[nodeIndex].nVoxelsPSum[j] = nVoxelsPSum[j];
				}

				__threadfence();

				atomicExch( &lpBuffer[location], nodeIndex | LP_OCCUPIED_BIT );

				done = true;
			}
			else if( v == LP_LOCK ) // someone is locking it
			{
				i--;
				continue; // try again
			}
			else
			{
				uint32_t otherNodeIndex = v & LP_VALUE_BIT;
				bool isEqual = outputOctreeNodes[otherNodeIndex].mask == mask;
				if( isEqual )
				for( int j = 0; j < 8; j++ )
				{
					if( outputOctreeNodes[otherNodeIndex].children[j] != children[j] )
					{
						isEqual = false;
						break;
					}
				}
				if( isEqual )
				{
					nodeIndex = otherNodeIndex;

					done = true;
				}
			}
		}

		if( d != -1 )
		{
			uint64_t mortonParent = inputOctreeTasks[itemIndex].getMortonParent();
			outputOctreeTasks[d].morton = mortonParent;
			outputOctreeTasks[d].child = nodeIndex;
			outputOctreeTasks[d].numberOfVoxels = numberOfVoxels;
		}
#endif
	}

    if( iteration % 2 == 0 )
	{
		if( threadIdx.x == 0 && blockIdx.x == 0 )
			g_octreeIterator1 = 0;
	}
	else
	{
		if( threadIdx.x == 0 && blockIdx.x == 0 )
			g_octreeIterator0 = 0;
	}
}

extern "C" __global__ void render( 
	uchar4* frameBuffer, int2 resolution, 
	uint32_t* taskCounter, StackElement* stackBuffer,
	CameraPinhole pinhole,
	IntersectorOctreeGPU intersector,
	int showVertexColor )
{
	__shared__ uint32_t taskIdx;

	StackElement* stack = stackBuffer + blockIdx.x * 32 * blockDim.x + threadIdx.x * 32;

	for (;; )
	{
		if( threadIdx.x == 0 )
		{
			taskIdx = atomicInc( taskCounter, 0xFFFFFFFF );
		}
		__syncthreads();

		uint32_t pixelIdx = taskIdx * blockDim.x + threadIdx.x;
		if( resolution.x * resolution.y <= pixelIdx )
		{
			break;
		}

		uint32_t x = pixelIdx % resolution.x;
		uint32_t y = pixelIdx / resolution.x;

		float3 ro, rd;
		pinhole.shoot( &ro, &rd, x, y, 0.5f, 0.5f, resolution.x, resolution.y );

		float t = MAX_FLOAT;
		int nMajor;
		uint32_t vIndex = 0;
		intersector.intersect( stack, ro, rd, &t, &nMajor, &vIndex );
		uchar4 colorOut = { 0, 0, 0, 255 };
		if( t != MAX_FLOAT )
		{
			if( showVertexColor )
			{
				colorOut = intersector.getVoxelColor( vIndex );
			}
			else
			{
				float3 hitN = getHitN( nMajor, rd );
				float3 color = ( hitN + float3{ 1.0f, 1.0f, 1.0f } ) * 0.5f;
				colorOut = { 255 * color.x + 0.5f, 255 * color.y + 0.5f, 255 * color.z + 0.5f, 255 };
			}
		}
		frameBuffer[y * resolution.x + x] = colorOut;
	}
}

extern "C" __global__ void HDRIstoreImportance( const float4* pixels, int2 resolution, double *sat, int cosWeighted, float3 axis )
{
	uint32_t pixelX = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t pixelY = blockIdx.y * blockDim.y + threadIdx.y;
	if( resolution.x <= pixelX || resolution.y <= pixelY )
	{
		return;
	}

	uint32_t pixelIdx = pixelY * resolution.x + pixelX;
	float dTheta = PI / (float)resolution.y;
	float dPhi = 2.0f * PI / (float)resolution.x;
	float theta = pixelY * dTheta;

	// dH = cos( theta ) - cos( theta + dTheta )
	//    = 2 sin( dTheta / 2 ) sin( dTheta / 2 + theta )
	float dH = 2.0f * INTRIN_SIN( dTheta * 0.5f ) * INTRIN_SIN( dTheta * 0.5f + theta );
	float dW = dPhi;
	float sr = dH * dW;
	float4 color = pixels[pixelIdx];

	float w = 1.0f;
	if( cosWeighted )
	{
		float sY = mix( INTRIN_COS( theta ), INTRIN_COS( theta + dTheta ), 0.5f );
		float phi = dPhi * ( (float)pixelX + 0.5f ) + PI;
		float sX = INTRIN_COS( phi );
		float sZ = INTRIN_SIN( phi );

		float sinTheta = INTRIN_SQRT( ss_max( 1.0f - sY * sY, 0.0f ) );
		float3 dirCenter = {
			sX * sinTheta,
			sY,
			sZ * sinTheta,
		};
		w = ss_max( dot( axis, dirCenter ), 0.0f );
	}

	sat[pixelIdx] = ( 0.2126f * color.x + 0.7152 * color.y + 0.0722 * color.z ) * sr * w;
}

template <class T, int NThreads>
__device__ inline T prefixSumInclusive( T prefix, T* sMemIO )
{
	for( uint32_t offset = 1; offset < NThreads; offset <<= 1 )
	{
		T x = sMemIO[threadIdx.x];

		if( offset <= threadIdx.x )
		{
			x += sMemIO[threadIdx.x - offset];
		}

		__syncthreads();

		sMemIO[threadIdx.x] = x;

		__syncthreads();
	}
	T sum = sMemIO[NThreads - 1];

	__syncthreads();

	sMemIO[threadIdx.x] += prefix;

	__syncthreads();

	return sum;
}

#define SAT_BLOCK_SIZE 512

extern "C" __global__ void buildSATh( int2 resolution, double* sat )
{
	__shared__ double s_mem[SAT_BLOCK_SIZE];
	int Y = blockIdx.x;

	double prefix = 0.0;
	for( int i = 0; i < resolution.x; i += SAT_BLOCK_SIZE )
	{
		int X = i + threadIdx.x;
		s_mem[threadIdx.x] = X < resolution.x ? sat[Y * resolution.x + X] : 0.0;

		__syncthreads();

		prefix += prefixSumInclusive<double, SAT_BLOCK_SIZE>( prefix, s_mem );

		if( X < resolution.x )
		{
			sat[Y * resolution.x + X] = s_mem[threadIdx.x];
		}
	}
}
extern "C" __global__ void buildSATv( int2 resolution, double* sat )
{
	__shared__ double s_mem[SAT_BLOCK_SIZE];
	int X = blockIdx.x;

	double prefix = 0.0;
	for( int i = 0; i < resolution.y; i += SAT_BLOCK_SIZE )
	{
		int Y = i + threadIdx.x;
		s_mem[threadIdx.x] = Y < resolution.y ? sat[Y * resolution.x + X] : 0.0;

		__syncthreads();

		prefix += prefixSumInclusive<double, SAT_BLOCK_SIZE>( prefix, s_mem );

		if( Y < resolution.y )
		{
			sat[Y * resolution.x + X] = s_mem[threadIdx.x];
		}
	}
}

extern "C" __global__ void buildSAT2u32( uint32_t* satU32, double* satF64, int n )
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	double sum = satF64[n - 1];
	if( i < n )
	{
		satU32[i] = (uint32_t)( satF64[i] / ( sum ) * (double)0xFFFFFFFFu );
	}
}

#define USE_PMJ 1
#define RENDER_NUMBER_OF_THREAD 64

extern "C" __global__ void renderPT(
	int iteration,
	float4* frameBuffer, int2 resolution,
	CameraPinhole pinhole,
	IntersectorOctreeGPU intersector,
	DynamicAllocatorGPU<StackElement> stackAllocator,
	HDRI hdri,
	PMJSampler pmj )
{
	uint32_t stackHandle;
	StackElement* stack = stackAllocator.acquire( &stackHandle );

	__shared__ uint32_t s_iterator;
	if( threadIdx.x == 0 )
	{
		s_iterator = 0;
	}
	__shared__ float localPixelValueXs[RENDER_NUMBER_OF_THREAD];
	__shared__ float localPixelValueYs[RENDER_NUMBER_OF_THREAD];
	__shared__ float localPixelValueZs[RENDER_NUMBER_OF_THREAD];
	localPixelValueXs[threadIdx.x] = 0.0f;
	localPixelValueYs[threadIdx.x] = 0.0f;
	localPixelValueZs[threadIdx.x] = 0.0f;

	__syncthreads();

	const int nBatchSpp = 16;

	for( int i = 0; i < nBatchSpp; i++ )
	{
		uint32_t taskIdx = atomicInc( &s_iterator, 0xFFFFFFFF );
		uint32_t localPixel = taskIdx / nBatchSpp;
		uint32_t localSpp = taskIdx % nBatchSpp;
		uint32_t pixelIdx = blockIdx.x * blockDim.x + localPixel;
		uint32_t spp = iteration * nBatchSpp + localSpp;

		if( pixelIdx < resolution.x * resolution.y )
		{
			uint32_t x = pixelIdx % resolution.x;
			uint32_t y = pixelIdx / resolution.x;

			MurmurHash32 hash( 0 );
			hash.combine( pixelIdx );
			uint32_t stream = hash.getHash();

#if defined( USE_PMJ )
			int dim = 0;
#define SAMPLE_2D() pmj.sample2d( spp, dim++, stream )
#else
			hash.combine( spp );
			PCG32 rng;
			rng.setup( 0, hash.getHash() );
#define SAMPLE_2D() float2{ uniformf( rng.nextU32() ), uniformf( rng.nextU32() ) }
#endif

			float2 cam_u01 = SAMPLE_2D();
			float3 ro, rd;
			pinhole.shoot( &ro, &rd, x, y, cam_u01.x, cam_u01.y, resolution.x, resolution.y );

			float3 T = { 1.0f, 1.0f, 1.0f };
			float3 L = {};
			for( int depth = 0; depth < 8; depth++ )
			{
				float t = MAX_FLOAT;
				int nMajor;
				uint32_t vIndex = 0;
				intersector.intersect( stack, ro, rd, &t, &nMajor, &vIndex );

				if( t == MAX_FLOAT )
				{
					//float I = ss_max( normalize( rd ).y, 0.0f ) * 3.0f;
					//float3 env = { I, I, I };
					if (depth == 0)
					{
						float3 env = hdri.sampleNearest( rd );
						L += T * env;
					}
					break;
				}

				float3 R = linearReflectance( intersector.getVoxelColor( vIndex ) );
				float3 hitN = getHitN( nMajor, rd );
				float3 hitP = ro + rd * t;

				{ // Explicit
					float2 u01 = SAMPLE_2D();
					float2 u23 = SAMPLE_2D();

					float3 dir;
					float3 emissive;
					float p;
					hdri.importanceSample( &dir, &emissive, &p, hitN, true, u01.x, u01.y, u23.x, u23.y );

					// no self intersection
					float t = MAX_FLOAT;
					int nMajor;
					uint32_t vIndex = 0;
					intersector.intersect( stack, hitP, dir, &t, &nMajor, &vIndex );
					if( t == MAX_FLOAT )
					{
						L += T * ( R / PI ) * ss_max( dot( hitN, dir ), 0.0f ) * emissive / p;
					}
				}

				T *= R;
			
				float2 u01 = SAMPLE_2D();
				float3 dir = sampleLambertian( u01.x, u01.y, hitN );

				ro = hitP; // no self intersection
				rd = dir;
			}

#undef SAMPLE_2D

			atomicAdd( &localPixelValueXs[localPixel], L.x );
			atomicAdd( &localPixelValueYs[localPixel], L.y );
			atomicAdd( &localPixelValueZs[localPixel], L.z );
		}
	}

	__syncthreads();

	uint32_t pixelIdx = blockIdx.x * blockDim.x + threadIdx.x;
	atomicAdd( &frameBuffer[pixelIdx].x, localPixelValueXs[threadIdx.x] );
	atomicAdd( &frameBuffer[pixelIdx].y, localPixelValueYs[threadIdx.x] );
	atomicAdd( &frameBuffer[pixelIdx].z, localPixelValueZs[threadIdx.x] );
	atomicAdd( &frameBuffer[pixelIdx].w, (float)nBatchSpp );
	stackAllocator.release( stackHandle );
}

extern "C" __global__ void renderResolve( uchar4* frameBufferU8, const float4* frameBufferF32, int n )
{
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if( i < n )
	{
		float4 value = frameBufferF32[i];
		uchar4 colorOut = {
			255 * INTRIN_POW( ss_min( value.x / value.w, 1.0f ), 1.0f / 2.2f ) + 0.5f,
			255 * INTRIN_POW( ss_min( value.y / value.w, 1.0f ), 1.0f / 2.2f ) + 0.5f,
			255 * INTRIN_POW( ss_min( value.z / value.w, 1.0f ), 1.0f / 2.2f ) + 0.5f,
			255 };
		frameBufferU8[i] = colorOut;
	}
}